// Copyright 2023 Ryan Moore
//
// 'Generic' is the enemy of 'Efficient'
//
// Tim Zaman


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() { printf("Hello, GPU!\n"); }

extern "C" void hello() {
  helloFromGPU<<<1, 1>>>();
  hipDeviceSynchronize();
}
